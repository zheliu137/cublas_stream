//Example 1. Application Using C and cuBLAS: 1-based indexing
//-----------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "hipblas.h"
//#include "cuda_settings.h"
#define IDX2F(i,j,ld) ((((j))*(ld))+((i)))

// void run_eig_wrapper_(const int N, cuDoubleComplex *x);
//void print_matrix(const int &m, const int &n, const cuDoubleComplex *A, const int &lda);
int  cusolver_c_stream(int N,hipDoubleComplex *A,int nmat);
void createRandoms(int size, double *randomArray);
//void cusolver(int N); 
//void test(); 

int main (int argc, char* argv[]){
    hipDoubleComplex *A;
    int N=10;
    if (argc > 1 ){
      N = strtol(argv[1],nullptr,0);
    }
    int nmat = 1;
    A = (hipDoubleComplex *)malloc(pow(N,2)*sizeof(hipDoubleComplex)*nmat);
    //A = (double *)malloc(pow(N,2)*sizeof(double));
    int size=N; 
    double *rand1;
    double *rand2;
    /* 
    rand1 = (double *)malloc(pow(size,2)*sizeof(double));
    rand2 = (double *)malloc(pow(size,2)*sizeof(double));
    printf("Generating %d by %d random matrix... \n",N,N);
    for (int l=0;l<nmat;l++){
    createRandoms(N, rand1);
    createRandoms(N, rand2);
    for (int i=0;i<N;i++){
    for (int j=0;j<N;j++){
      A[IDX2F(i,j,N)+l*N*N] = {rand1[i+j*N]+rand1[j+i*N],rand2[i+j*N]-rand2[j+i*N]};
    }
    } 
    } 
    */
    for (int i=0;i<N;i++){
    for (int j=0;j<N;j++){
      //A[IDX2F(i,j,N)] = {double(i*j+1), double(3*i*j*(i-j)-j+i)};
      A[IDX2F(i,j,N)] = {double(i+j+1.0), 10.0*(i-j)};
      //A[IDX2F(i,j,N)] = float(i*j);
      //A[IDX2F(i,j,N)] = {1.0,1.2};
      //printf("%f\n",A[IDX2F(i,j,N)]);
    }
    }

    printf("Success.\n");

    cusolver_c_stream( N, A, nmat);

    }

#include <cstdio>
#include <cstdlib>
#include <vector>
#include <complex>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

//#define DEBUG
#define SINGLERUN
#define nstream 10
#ifdef DEBUG
#define CUSOLVER_CHECK(err) (HandlecusolverError(err, __FILE__, __LINE__))
#define CUDA_CHECK(err) (HandleError(err, __FILE__, __LINE__))
#else
#define CUSOLVER_CHECK(err) (err)
#define CUDA_CHECK(err) (err)
#endif

static void HandlecusolverError(hipsolverStatus_t err, const char *file, int line )
{

    if (err != HIPSOLVER_STATUS_SUCCESS)
    {
        fprintf(stderr, "ERROR: %d in %s at line %d, (error-code %d)\n",
                err, file, line, err);
        fflush(stdout);
        exit(-1);
    }
}

static void HandleError(hipError_t err, const char *file, int line)
{

    if (err != hipSuccess)
    {
        fprintf(stderr, "ERROR: %s in %s at line %d (error-code %d)\n",
                hipGetErrorString(err), file, line, err);
        fflush(stdout);
        exit(-1);
    }
}

template <typename T> void print_matrix(const int &m, const int &n, const T *A, const int &lda);

template <> void print_matrix(const int &m, const int &n, const float *A, const int &lda) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            std::printf("%0.2f ", A[j * lda + i]);
        }
        std::printf("\n");
    }
}

template <> void print_matrix(const int &m, const int &n, const double *A, const int &lda) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            std::printf("%0.2f ", A[j * lda + i]);
        }
        std::printf("\n");
    }
}

template <> void print_matrix(const int &m, const int &n, const hipComplex *A, const int &lda) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            std::printf("%0.2f + %0.2fj ", A[j * lda + i].x, A[j * lda + i].y);
        }
        std::printf("\n");
    }
}

template <>
void print_matrix(const int &m, const int &n, const hipDoubleComplex *A, const int &lda) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            std::printf("%0.2f + %0.2fj ", A[j * lda + i].x, A[j * lda + i].y);
        }
        std::printf("\n");
    }
}

//extern "C"
//{
int cusolver_c_stream(const int m,  hipDoubleComplex *A_, const int nmat_ ) {

    const int lda = m;
    const int nmat = 30;

    hipsolverHandle_t cusolverH[nstream];
    hipStream_t stream[nstream];
    hipsolverSyevjInfo_t syevj_params[nstream];

    printf("solving %d %dx%d matrices by Jacobi method with %d streams.\n",nmat,m,m, nstream);
    hipDoubleComplex *A; // matrix should be stored in pinned memory

    CUDA_CHECK(hipHostMalloc((void **)&A,sizeof(hipDoubleComplex)*lda * m * nmat, hipHostMallocDefault));
    //A = (cuDoubleComplex *)malloc (m*lda * nmat * sizeof (cuDoubleComplex));

    hipDoubleComplex *V; // eigenvectors
    double *W; // eigenvalue
    hipDoubleComplex *AMV; // A*V
    AMV = (hipDoubleComplex *)malloc (m * nmat * sizeof (*AMV));
    //V = (cuDoubleComplex *)malloc (m*lda * nmat * sizeof (*V));
    //W = (double *)malloc (m * nmat * sizeof (double));
    CUDA_CHECK(hipHostMalloc((void **)&V,sizeof(hipDoubleComplex)*lda * m * nmat, hipHostMallocDefault));
    CUDA_CHECK(hipHostMalloc((void **)&W,sizeof(double) * m * nmat, hipHostMallocDefault));

    // copy to pinned memory
    printf("Copy matrix to pinned memory.\n");
    for (int i=0;i<nmat;i++) {
      std::copy(A_,A_+lda*m,A+i*lda*m);
    }

    hipDoubleComplex *d_A;
    double *d_W;
    int *devInfo;
    hipDoubleComplex *d_work[nstream];
    int lwork[nstream];
    int info_gpu[nmat];

    /* configuration of syevj  */
    const double tol = 1.e-10;
    const int max_sweeps = 15;
    const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvectors.
    const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    // step 0: allocate device memory
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(hipDoubleComplex) * lda * m * nmat));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_W), sizeof(double) * m * nstream));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&devInfo), sizeof(int)*nstream));


    for (int i=0; i < nstream; i++ ) {
       int ist = i;
    /* step 1: create cusolver handle, bind a stream */
      CUDA_CHECK(hipStreamCreateWithFlags(&stream[i], hipStreamNonBlocking));


   }
    /* step 3: copy A to device */
    CUDA_CHECK(
        hipMemcpy(d_A, A, sizeof(hipDoubleComplex) * lda * m * nmat, hipMemcpyHostToDevice ));
    /* step 4: query working space of syevj */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH[0]));
    CUSOLVER_CHECK(
          hipsolverDnZheevj_bufferSize(cusolverH[0], jobz, uplo, m, 
          &d_A[0], lda, &d_W[0], &lwork[0], syevj_params[0]));
    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH[0]));

    for (int i=0; i < nstream; i++ ) {
       int ist = i;
       CUDA_CHECK(hipMallocAsync(reinterpret_cast<void **>(&d_work[ist]), 
                          sizeof(hipDoubleComplex) * lwork[0],stream[ist]));
    }

    int nloop = 1  ;
    int nnn = 5000;
    for (int l=0; l < nloop; l++ ) {
    printf("loop\n");
    //for (int i=0; i < nmat; i++ ) {
    for (int i=0; i < nnn; i++ ) {
    //printf("matrices %d\n",i);
    int ist = i%nstream;
    printf("stream =  %d\n",ist);
    //int ist = 0;
    // printf("start %dth matrix, stream = %u \n", i, stream[ist]);

    //printf("1.1\n");
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH[ist]));

    //printf("1.2\n");
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH[ist], stream[ist]));
    //printf("1.3\n");
    /* step 2: configuration of syevj */
    CUSOLVER_CHECK(hipsolverDnCreateSyevjInfo(&syevj_params[ist]));

    /* default value of tolerance is machine zero */
    CUSOLVER_CHECK(hipsolverDnXsyevjSetTolerance(syevj_params[ist], tol));

    /* default value of max. sweeps is 100 */
    CUSOLVER_CHECK(hipsolverDnXsyevjSetMaxSweeps(syevj_params[ist], max_sweeps));


    /* step 5: compute eigen-pair   */
    CUSOLVER_CHECK(hipsolverDnZheevj(cusolverH[ist], jobz, uplo, m, 
                                    &d_A[ist*lda*m], lda, &d_W[ist*m], 
                                    d_work[ist], lwork[0], &devInfo[ist],
                                    syevj_params[ist]));

    //printf("1.4\n");
    //CUDA_CHECK(cudaMemcpyAsync(&V[lda*m*i], &d_A[i*m*lda], 
    //       sizeof(cuDoubleComplex) * lda * m, cudaMemcpyDeviceToHost, stream[ist]));
    //CUDA_CHECK(cudaMemcpyAsync(&W[m*i], &d_W[ist*m], 
    //       sizeof(double) * m, cudaMemcpyDeviceToHost, stream[ist]));

    //CUDA_CHECK(cudaFreeAsync(d_work[ist],stream[ist]));
    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH[ist]));
    //printf("done. %d\n",i);
    CUDA_CHECK(hipDeviceSynchronize());
    }
    }

    CUDA_CHECK(hipMemcpyAsync(&info_gpu[0], &devInfo[0], 
           sizeof(int), hipMemcpyDeviceToHost, stream[0]));
    printf("%d",info_gpu[0]);
    CUDA_CHECK(hipDeviceSynchronize());
    /*
    // step 6: check results
    double residual;
    for (int i=0; i < nmat; i++ ) {
       residual = 0.0;
       for (int j=0;j < m; j++) { 
#ifdef DEBUG
           printf("A * V(%d), W(%d) * V (%d)\n",j,j,j);
#endif
           for (int k=0; k < m; k++) { 
               AMV[k] = {0.0,0.0};
               for (int l=0; l < m; l++) { 
                   AMV[k] = cuCadd(AMV[k],
                            cuCmul(A_[k+l*m+i*m*lda], V[i*m*lda+l+j*m]));
               }
#ifdef DEBUG
               printf("%0.2f + %0.2fj ", AMV[k].x, AMV[k].y);
               printf("%0.2f + %0.2fj ", 
                     W[i*m+j]*V[i*m*lda+k+j*m].x, W[i*m+j]*V[i*m*lda+k+j*m].y);
               printf("\n");
#endif
               residual = residual + abs(AMV[k].x-W[i*m+j]*V[i*m*lda+k+j*m].x)+
                                     abs(AMV[k].y-W[i*m+j]*V[i*m*lda+k+j*m].y);
           }
       }
    }

#ifdef SINGLERUN
    printf("residual = %e \n", residual);
#endif
    std::copy(V,V+lda*m*nmat,A_);
    */
    // step 7 free device memory and reset device

    CUDA_CHECK(hipHostFree(A));
    CUDA_CHECK(hipHostFree(W));
    CUDA_CHECK(hipHostFree(V));

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_W));
    CUDA_CHECK(hipFree(devInfo));

    for (int i=0; i < nstream; i++ ) {
      CUDA_CHECK(hipStreamDestroy(stream[i]));
      CUSOLVER_CHECK(hipsolverDnDestroySyevjInfo(syevj_params[i]));
    //  CUSOLVER_CHECK(cusolverDnDestroy(cusolverH[i]));
    }

    CUDA_CHECK(hipDeviceReset());


    return EXIT_SUCCESS;
}
